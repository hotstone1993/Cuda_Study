#include "hip/hip_runtime.h"
#include "cuda_support.h"
#include "00_MatMul.cuh"
#include "00_MatMul_Const.h"

__global__ void mulMatrix(TARGET_TYPE* c, const TARGET_TYPE* a, const TARGET_TYPE* b, const unsigned int N)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (row >= N || col >= N)
        return;
    
    TARGET_TYPE sum = 0;

    for (unsigned int idx = 0; idx < N; ++idx) {
        sum += (a[N * row + idx] * b[idx * N + col]);
    }
    c[row * N + col] = sum;
}


__global__ void mulMatrixWithSharedMemory(TARGET_TYPE* c, const TARGET_TYPE* a, const TARGET_TYPE* b, const unsigned int N)
{
    __shared__ TARGET_TYPE tempA[THREADS][THREADS];
    __shared__ TARGET_TYPE tempB[THREADS][THREADS];

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int localCol = threadIdx.x;
    int localRow = threadIdx.y;

    TARGET_TYPE sum = 0;

    for (unsigned int bid = 0; bid < ceil((float)N / blockDim.x); ++bid) {
        if (row < N && bid * blockDim.x + localCol < N) {
            tempA[localRow][localCol] = a[N * row + (bid * blockDim.x + localCol)];
        }
        else {
            tempA[localRow][localCol] = 0;
        }

        if (col < N && bid * blockDim.y + localRow < N) {
            tempB[localRow][localCol] = b[N * (bid * blockDim.y + localRow) + col];
        }
        else {
            tempB[localRow][localCol] = 0;
        }

        __syncthreads();

        for (unsigned int idx = 0; idx < blockDim.x; ++idx) {
            sum += (tempA[localRow][idx] * tempB[idx][localCol]);
        }

        __syncthreads();
    }

    if (row >= N || col >= N)
        return;

    c[row * N + col] += sum;
}

template <class T1>
void initRandom(std::vector<T1*>& inputs) {
    srand(static_cast<unsigned int>(time(nullptr)));

    for (size_t idx = 0; idx < SIZE * SIZE; ++idx) {
        inputs[HOST_INPUT1][idx] = rand() % 21 - 10;
        inputs[HOST_INPUT2][idx] = rand() % 21 - 10;
    }
}

template <class T1>
void copyInputs(std::vector<T1*>& inputs) {
    hipError_t cudaStatus = hipMemcpy(inputs[DEVICE_INPUT1], inputs[HOST_INPUT1], SIZE * SIZE * sizeof(T1), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("hipMemcpy failed! (Host to Device)");
    }
    cudaStatus = hipMemcpy(inputs[DEVICE_INPUT2], inputs[HOST_INPUT2], SIZE * SIZE * sizeof(T1), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("hipMemcpy failed! (Host to Device)");
    }
}

template <class T1, class T2>
void basic::setup(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    inputs.resize(INPUT_COUNT);
    outputs.resize(OUTPUT_COUNT);

    inputs[HOST_INPUT1] = new T1[SIZE * SIZE];
    inputs[HOST_INPUT2] = new T1[SIZE * SIZE];
    CUDA_MALLOC(inputs[DEVICE_INPUT1], SIZE * SIZE, T1)
    CUDA_MALLOC(inputs[DEVICE_INPUT2], SIZE * SIZE, T1)

    outputs[HOST_OUTPUT1] = new T2[SIZE * SIZE];
    CUDA_MALLOC(outputs[DEVICE_OUTPUT1], SIZE * SIZE, T2)

    initRandom(inputs);
    copyInputs(inputs);
}

template <class T1, class T2>
void basic::destroy(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    delete[] inputs[HOST_INPUT1];
    delete[] inputs[HOST_INPUT2];
    hipFree(inputs[DEVICE_INPUT1]);
    hipFree(inputs[DEVICE_INPUT2]);

    delete[] outputs[HOST_OUTPUT1];
    hipFree(outputs[DEVICE_OUTPUT1]);
}

template <class T1, class T2>
void basic::run(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    int face = (SIZE + THREADS - 1) / THREADS;
    dim3 gridDim(face, face);
    dim3 blockDim(THREADS, THREADS);

    mulMatrixWithSharedMemory<<<gridDim, blockDim>>>(outputs[DEVICE_OUTPUT1], inputs[DEVICE_INPUT1], inputs[DEVICE_INPUT2], SIZE);
    hipDeviceSynchronize();

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("MatMul launch failed: %s\n");
    }
    
    cudaStatus = hipMemcpy(outputs[HOST_OUTPUT1], outputs[DEVICE_OUTPUT1], SIZE * SIZE * sizeof(T2), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("hipMemcpy failed! (Device to Host)");
    }
}

template void basic::setup(std::vector<TARGET_TYPE*>& inputs, std::vector<TARGET_TYPE*>& outputs);
template void basic::destroy(std::vector<TARGET_TYPE*>& inputs, std::vector<TARGET_TYPE*>& outputs);
template void basic::run(std::vector<TARGET_TYPE*>& inputs, std::vector<TARGET_TYPE*>& outputs);