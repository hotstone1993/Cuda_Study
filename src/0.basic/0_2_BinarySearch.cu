#include "hip/hip_runtime.h"
#include "0_2_BinarySearch.cuh"

namespace cg = cooperative_groups;

__global__ void binarySearch() {
    
}

template <class T1, class T2>
void basic::binary_search::run(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    dim3 gridDim(SIZE - (THREADS - 1) / THREADS);
    dim3 blockDim(THREADS);

    checkCudaError(hipMemcpy(inputs[HOST_INPUT], outputs[DEVICE_OUTPUT], SIZE * sizeof(T2), hipMemcpyDeviceToHost), "hipMemcpy failed! (Device to Host) - ");
}

template void basic::binary_search::run(std::vector<TARGET_INPUT_TYPE*>& inputs, std::vector<TARGET_OUTPUT_TYPE*>& outputs);