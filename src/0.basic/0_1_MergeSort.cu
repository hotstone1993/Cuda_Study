#include "hip/hip_runtime.h"
#include "0_1_MergeSort.cuh"

__global__ void mergeSort(TARGET_INPUT_TYPE* input)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    printf("I am thread %d, my SM ID is %d, my warp ID is %d, and my warp lane is %d\n",
          idx, __mysmid(), __mywarpid(), __mylaneid());
}

template <class T1, class T2>
void basic::merge::run(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    dim3 gridDim(SIZE - (THREADS - 1) / THREADS);
    dim3 blockDim(THREADS);

    mergeSort<<<gridDim, blockDim>>>(inputs[DEVICE_INPUT]);

    checkCudaError(hipMemcpy(inputs[HOST_INPUT], inputs[DEVICE_INPUT], SIZE * sizeof(T2), hipMemcpyDeviceToHost), "hipMemcpy failed! (Device to Host) - ");
}

template void basic::merge::run(std::vector<TARGET_INPUT_TYPE*>& inputs, std::vector<TARGET_OUTPUT_TYPE*>& outputs);