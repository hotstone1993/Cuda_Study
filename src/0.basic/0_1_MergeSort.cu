#include "0_1_MergeSort.cuh"

template <class T1, class T2>
void basic::merge::run(std::vector<T1*>& inputs, std::vector<T2*>& outputs) {
    // checkCudaError(hipMemcpy(outputs[HOST_OUTPUT1], outputs[DEVICE_OUTPUT1], SIZE * SIZE * sizeof(T2), hipMemcpyDeviceToHost), "hipMemcpy failed! (Device to Host) - ");
}

template void basic::merge::run(std::vector<TARGET_INPUT_TYPE*>& inputs, std::vector<TARGET_OUTPUT_TYPE*>& outputs);